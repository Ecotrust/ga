
#include <hip/hip_runtime.h>
/* This file allocates memory on GPU device/s */
#if HAVE_STDIO_H
#   include <stdio.h>
#endif

extern "C" void gpu_mem_alloc(int *d_a, int d_size) {

  hipMalloc((void **)&d_a, d_size*sizeof(int));
  // return 1;  // replace with CUDA_ERROR code
}
